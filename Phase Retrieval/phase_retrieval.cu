#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include"tiffio.h"
#include<cmath>
#include<algorithm>
#include"FT_kernel.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include<hip/hip_runtime.h>
#include"myUnwrap.h"
#include"matrix.h"


#define lambda 0.632
#define pi 3.1415926
#define TILE_WIDTH 16

typedef unsigned char BYTE;

using namespace std;

class image {
public:
	char filename[50];
	float maxIntensity, minIntensity;
	int height, width;
	int imagePixels;
	uint8** rawImage;
	float2* imageData;
	int2 fftMaxPosition;
	float2* filteredBaseband;
	float* imageFilter;

	image() {
		rawImage = (uint8 * *)malloc(sizeof(uint8*) * 960);
		for (int i = 0; i < 960; i++) {
			rawImage[i] = (uint8*)malloc(sizeof(uint8) * 1280);
		}
		filteredBaseband = (float2*)malloc(sizeof(float2) * 1280 * 960);
		imageFilter = new float[960 * 1280];
	}
};

bool getImageInfo(image* targetImage);// Used for get the useful data in tiff image, and return false when the filename of the target image is invalid.
float* phaseRetrieval(image* calibImage, image* testImage);
void fourierFilterForCalib(image* calibImage);
int2 findMaxPoint(float* input);
void imageFileWrite(float* input, char* filename);
void complexWrite(const char* title, float2* input, int width, const char* filename);
void realWrite(const char* title, float* input, int width, const char* filename);
void errorHandle(int input);
float* phaseUnwrapping(float* wMatrix);

__global__ void vectorAdd(float* a, float* b, float* c, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements)
		c[i] = a[i] + b[i];
}

__global__ void vectorNumMultiple(float* a, float* b, float* c, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements)
		c[i] = a[i] * b[i];
}

__global__ void numMultipleForComplex(float2* a, float* b, float2* c, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x + threadIdx.y * gridDim.x;
	if (i < numElements) {
		c[i].x = a[i].x * b[i];
		c[i].y = a[i].y * b[i];
	}
}

__global__ void vectorMatDivide(float2* dividend, float2* divisor, float2* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		output[i].x = dividend[i].x / divisor[i].x;
		output[i].y = dividend[i].y / divisor[i].y;
	}
}


__global__ void vectorNumdivide(float2* dividend, int divisor, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		dividend[i].x = dividend[i].x / divisor;
		dividend[i].y = dividend[i].y / divisor;
	}
}

__global__ void getAbsOfComplexMatric(hipfftComplex* input, hipfftReal* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements)
		output[i] = sqrt(input[i].x * input[i].x + input[i].y * input[i].y);
}

__global__ void createFilter(int padding, int2 maxPoint, float2* input, float2* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		if ((x - 639) * (x - 639) + (y - 479) * (y - 479) <= padding * padding) {
			output[i].x = input[i].x;
			output[i].y = input[i].y;
		}
		else {
			output[i].x = 0;
			output[i].y = 0;
		}
	}
}

__global__ void FFTShift2D(hipfftComplex* input, hipfftComplex* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	int halfX = 640;
	int halfY = 480;
	if (i < numElements) {
		if (y < halfY) {
			if (x < halfX) {
				output[i].x = input[(x + halfX) * blockDim.y + y + halfY].x;
				output[i].y = input[(x + halfX) * blockDim.y + y + halfY].y;
			}
			else {
				output[i].x = input[(x - halfX) * blockDim.y + y + halfY].x;
				output[i].y = input[(x - halfX) * blockDim.y + y + halfY].y;
			}
		}
		else {
			if (x < halfX) {
				output[i].x = input[(x + halfX) * blockDim.y + y - halfY].x;
				output[i].y = input[(x + halfX) * blockDim.y + y - halfY].y;
			}
			else {
				output[i].x = input[(x - halfX) * blockDim.y + y - halfY].x;
				output[i].y = input[(x - halfX) * blockDim.y + y - halfY].y;
			}
		}
	}
}

__global__ void IFFTShift2D(hipfftComplex* input, hipfftComplex* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	int halfY = blockDim.y / 2;
	int halfX = gridDim.x / 2;
	int preX, preY;
	if (i < numElements) {
		if (y < halfY) {
			if (x < halfX) {
				output[i].x = input[(x + halfX) * blockDim.y + y + halfY].x;
				output[i].y = input[(x + halfX) * blockDim.y + y + halfY].y;
			}
			else {
				output[i].x = input[(x - halfX) * blockDim.y + y + halfY].x;
				output[i].y = input[(x - halfX) * blockDim.y + y + halfY].y;
			}
		}
		else {
			if (x < halfX) {
				output[i].x = input[(x + halfX) * blockDim.y + y - halfY].x;
				output[i].y = input[(x + halfX) * blockDim.y + y - halfY].y;
			}
			else {
				output[i].x = input[(x - halfX) * blockDim.y + y - halfY].x;
				output[i].y = input[(x - halfX) * blockDim.y + y - halfY].y;
			}
		}
	}
}

__global__ void circShift2D(hipfftComplex* input, int2 maxPoint, hipfftComplex* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	int preX = x - 639 + maxPoint.x;
	int preY = y - 479 + maxPoint.y;
	if (i < numElements) {
		/*
		if (preX < 0)
			preX = 1280 + preX;
		if (preX >= 1280)
			preX = preX - 1280;
		if (preY < 0 || preY > 480) {
			preX = 1279 - preX;
			if (preY < 0)
				preY = -preY;
			else
				preY = 480 - preY;
		}
		else {
			output[y + x * blockDim.y].x = input[preY + preX * 481].x;
			output[y + x * blockDim.y].y = input[preY + preX * 481].y;
		}
		*/
		if (preX < 0)
			preX = 1280 + preX;
		if (preX >= 1280)
			preX = preX - 1280;
		if (preY < 0)
			preY = 960 + preY;
		if (preY >= 960)
			preY = preY - 960;
		output[y + x * blockDim.y].x = input[preY + preX * blockDim.y].x;
		output[y + x * blockDim.y].y = input[preY + preX * blockDim.y].y;
	}
}

__global__ void phaseCalculate(hipfftComplex* input, hipfftReal* output, int numElements) {

	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		output[i] = atan2(input[i].y, input[i].x);
	}
}

__global__ void createXConfVec(float xConf, float vecStep, hipfftReal* output, int numElements) {
	int i = threadIdx.x;
	if (i < numElements) {
		output[i] = xConf - i * vecStep;
	}
}

__global__ void forPhaseImage(float mean2, hipfftReal* xConfVec, hipfftReal* input, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		input[x + y * gridDim.x] = input[x + y * gridDim.x] + xConfVec[x] - mean2;
		if (input[x + y * gridDim.x] < 0)
			input[x + y * gridDim.x] = -input[x + y * gridDim.x];
	}
}

__global__ void calHeight(hipfftReal* input, float mean2, hipfftReal* output, int numElements) {
	int dn = 0.075;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		output[i] = (input[i] * lambda / 2 / pi / dn - mean2) / 4 * 3;
	}
}

__global__ void calOutputImage(hipfftReal* input, float* output, int numElements) {
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		output[i] = ((input[i] + 15) / 30) * 256 - 1;
	}
}

__global__ void matrixMultiple(float* output, float* matrixL, float* matrixR , int size ,int numElements) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * (blockDim.y * gridDim.y) + (threadIdx.y + blockIdx.y * blockDim.y);
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockDim.y + threadIdx.y;
	if (i < numElements) {
		float data;
		for (int a = 0; a < size; a++) {
			data += matrixL[a * size + y] * matrixR[a + x * size];
		}
		output[i] = data;
	}
}

__global__ void DCTMatrixL(int height, float* DCTMatrixL) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * (blockDim.y * gridDim.y) + (threadIdx.y + blockIdx.y * blockDim.y);
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockDim.y + threadIdx.y;
	DCTMatrixL[i] = cospi((2 * x + 1) * y / 2 * height);
}

__global__ void DCTMatrixR(int width, float* DCTMatrixR) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * (blockDim.y * gridDim.y) + (threadIdx.y + blockIdx.y * blockDim.y);
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockDim.y + threadIdx.y;
	DCTMatrixR[i] = cospi((2 * y + 1) * x / 2 * width);
}

__global__ void matrixModify(float* input ,int width, int height) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * (blockDim.y * gridDim.y) + (threadIdx.y + blockIdx.y * blockDim.y);
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockDim.y + threadIdx.y;
	input[i] = input[i] / (2 * (cospi(x / width) + cospi(y / height) - 2));
}

__global__ void matrixModify(float* input, int width, int height) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * (blockDim.y * gridDim.y) + (threadIdx.y + blockIdx.y * blockDim.y);
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockDim.y + threadIdx.y;
	input[i] = input[i] / (2 * (cospi((float)x / width) + cospi((float)y / height) - 2));
	if (x == 0)
		if (y == 0)
			input[i] = input[i] / (height * width);
		else
			input[i] = input[i] * 2 / (height * width);
	else
		if (y == 0)
			input[i] = input[i] / height * width;
		else
			input[i] = input[i] * 2 / (height * width);
}

__global__ normalizer(float* input , int height , int width) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * (blockDim.y * gridDim.y) + (threadIdx.y + blockIdx.y * blockDim.y);
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockDim.y + threadIdx.y;
	if (x == 0)
		if (y == 0)
			input[i] = input[i] * sqrt((float)1 / (height * width));
		else
			input[i] = input[i] * sqrt((float)2 / (height * width));
	else
		if (y == 0)
			input[i] = input[i] / sqrt((float)height * width);
		else
			input[i] = input[i] * sqrt((float)2 / (height * width));
}

__global__ void gradCal(float* input, float* output, int height, int width) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * (blockDim.y * gridDim.y) + (threadIdx.y + blockIdx.y * blockDim.y);
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockDim.y + threadIdx.y;
	if (x < width - 1 && y < height - 1 && x > 0 && y > 0) {
		output[i] = input[i + height] + input[i - height] - 4 * input[i] + input[i + 1] + input[i - 1];
	}
	else if (x == 0) {
		if (y == 0)
			output[i] = input[i + height] - 4 * input[i] + input[i + 1];
		else if (y == width - 1)
			output[i] = input[i + height] - 4 * input[i] + input[i - 1];
		else
			output[i] = input[i + height] - 4 * input[i] + input[i + 1] + input[i - 1];
	}
	else if (x == width - 1) {
		if (y == 0)
			output[i] = input[i - height] - 4 * input[i] + input[i + 1];
		else if (y == width - 1)
			output[i] = input[i - height] - 4 * input[i] + input[i - 1];
		else
			output[i] = input[i - height] - 4 * input[i] + input[i + 1] + input[i - 1];
	}
}

/*
__global__ void MatrixMultiple(int m, int n, int k, float* A, float* B, float* C)
{
	__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x;		int by = blockIdx.y;
	int tx = threadIdx.x;		int ty = threadIdx.y;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	double Cvalue = 0;

	for (int t = 0; t < (n - 1) / TILE_WIDTH + 1; ++t)
	{
		
		if (Row < m && t * TILE_WIDTH + tx < n)		//Խ�紦�����������С�ľ�����ˣ���ѡ��
			//ds_A[tx][ty] = A[t*TILE_WIDTH + tx][Row];
			ds_A[tx][ty] = A[Row * n + t * TILE_WIDTH + tx];//�Ժϲ��ķ�ʽ������Ƭ
		else
			ds_A[tx][ty] = 0.0;

		if (t * TILE_WIDTH + ty < n && Col < k)
			//ds_B[tx][ty] = B[Col][t*TILE_WIDTH + ty];
			ds_B[tx][ty] = B[(t * TILE_WIDTH + ty) * k + Col];
		else
			ds_B[tx][ty] = 0.0;

		//��֤tile�����е�Ԫ�ر�����
		__syncthreads();

		for (int i = 0; i < TILE_WIDTH; ++i)
			Cvalue += ds_A[i][ty] * ds_B[tx][i];//��shared memory��ȡֵ

		//ȷ�������߳���ɼ���󣬽�����һ���׶εļ���
		__syncthreads();

		if (Row < m && Col < k)
			C[Row * k + Col] = Cvalue;
	}
}
*/


int main() {
	image calibImage, testImage;
	do
	{
		cout << "please input the filename and the address of the calibration image: ";
		cin >> calibImage.filename;
	} while (!getImageInfo(&calibImage));
	do
	{
		cout << "please input the filename and the address of the test image: ";
		cin >> testImage.filename;
	} while (!getImageInfo(&testImage));
	fourierFilterForCalib(&calibImage);
	float* result = phaseRetrieval(&calibImage, &testImage);
}

bool getImageInfo(image* targetImage) {

	TIFF* tif = TIFFOpen(targetImage->filename, "r");
	if (tif) {
		TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &targetImage->height);
		TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &targetImage->width);
		targetImage->imagePixels = targetImage->height * targetImage->width;
		targetImage->imageData = new float2[targetImage->imagePixels];
		targetImage->rawImage = new uint8 * [targetImage->height];
		for (int i = 0; i < targetImage->height; i++) {
			targetImage->rawImage[i] = (uint8*)malloc(sizeof(uint8) * targetImage->width);
		}
		for (int i = 0; i < targetImage->height; i++) {
			TIFFReadScanline(tif, targetImage->rawImage[i], i);
		}
		float tempMax = 0;
		float tempMin = 10000;
		for (int i = 0; i < targetImage->width; i++) {
			for (int j = 0; j < targetImage->height; j++) {
				targetImage->imageData[i * targetImage->height + j].x = ((float)targetImage->rawImage[j][i] / 255);
				targetImage->imageData[i * targetImage->height + j].y = 0;
				//if (tempMax < targetImage->imageData[i * targetImage->width + j]) 
				//	tempMax = targetImage->imageData[i * targetImage->width + j];
				//if (tempMin > targetImage->imageData[i * targetImage->width + j])
				//	tempMin = targetImage->imageData[i * targetImage->width + j];
			}
		}
		targetImage->maxIntensity = tempMax;
		targetImage->minIntensity = tempMin;
		TIFFClose(tif);
		return true;
	}
	else {
		cout << "File Open Error! please input a valid filename" << endl;
		return false;
	}

	return true;
}

int2 findMaxPoint(float* input) {
	int2 tempPoint = { 0,0 };
	float tempMax = 0;
	for (int i = 0; i < 960; i++) {
		for (int j = 700; j < 1280; j++) {
			if (input[i + j * 960] > tempMax) {
				tempMax = input[i + j * 960];
				tempPoint.y = i;
				tempPoint.x = j;
			}
		}
	}
	return tempPoint;
}

void fourierFilterForCalib(image* calibImage) {
	cout << "Part: fourier filter for calib image" << endl;


	hipEvent_t FFStart;
	hipEventCreate(&FFStart);
	hipEvent_t FFStop;
	hipEventCreate(&FFStop);
	hipEventRecord(FFStart, NULL);


	int imageSizeL = 1280 * 960;
	dim3 blockSizeL(1, 960, 1), gridSize(1280, 1, 1);
//	float2* tempComplex = new float2[imageSizeL];
	float* calibAbsImage = new float[imageSizeL];

	//complexWrite("input for fourierFiltered", calibImage->imageData, 960, "../Debug/input_FF.csv");

	hipfftReal* dev_calibABSFFTShifted;
	hipfftComplex* dev_calibFFT, * dev_circCalibFFT, * dev_calibFilteredBaseband, * dev_calibCircFilteredFFT, * dev_filteredCalibFFT, * dev_calibFFTShifted, * dev_calibImage;
	int n = hipMalloc((void**)& dev_calibImage, sizeof(float2) * calibImage->imagePixels);
//	if (hipSuccess != n)
//		cout << "cuda malloc error1!" << endl;
//	cout << n << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibFilteredBaseband, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error2!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibCircFilteredFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error3!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibABSFFTShifted, sizeof(float) * imageSizeL))
		cout << "cuda malloc error4!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error5!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_circCalibFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error8!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_filteredCalibFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error8!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibFFTShifted, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error8!" << endl;

	hipfftHandle FFT;
	errorHandle(hipfftPlan2d(&FFT, 1280, 960, HIPFFT_C2C));

	if (hipSuccess != hipMemcpy(dev_calibImage, calibImage->imageData, calibImage->imagePixels * sizeof(hipfftComplex), hipMemcpyHostToDevice))
		cout << "cuda memory cpy error!" << endl;

	hipEvent_t fftStart;
	hipEventCreate(&fftStart);
	hipEvent_t fftStop;
	hipEventCreate(&fftStop);
	hipEventRecord(fftStart, NULL);

	errorHandle(hipfftExecC2C(FFT, dev_calibImage, dev_calibFFT, HIPFFT_FORWARD));

	hipEventRecord(fftStop, NULL);
	hipEventSynchronize(fftStop);
	float msecFFT = 0.0f;
	hipEventElapsedTime(&msecFFT, fftStart, fftStop);
	cout << "total runtime of FFT: " << msecFFT << " ms" << endl;

//	float2* tempOut = (float2*)malloc(sizeof(float2) * 1280 * 960);
//	int a = hipMemcpy((void*)tempOut, (void*)dev_calibFFT, imageSizeL * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
//	if (hipSuccess != a)
//		cout << "cuda memory cpy error!" << endl;
//	cout << a << endl;
//	complexWrite("temp dubug info", tempOut, 960, "../Debug/calib_FFT.csv");

	hipEvent_t fftShiftStart;
	hipEventCreate(&fftShiftStart);
	hipEvent_t fftShiftStop;
	hipEventCreate(&fftShiftStop);
	hipEventRecord(fftShiftStart, NULL);

	FFTShift2D << < gridSize, blockSizeL >> > (dev_calibFFT, dev_calibFFTShifted, imageSizeL);

	hipEventRecord(fftShiftStop, NULL);
	hipEventSynchronize(fftShiftStop);
	float msecFFTShift = 0.0f;
	hipEventElapsedTime(&msecFFTShift, fftShiftStart, fftShiftStop);
	cout << "total runtime of FFT: " << msecFFTShift << " ms" << endl;

//	if (hipSuccess != hipGetLastError())
//		printf("FFTShift error!\n");
//	if (hipSuccess != hipMemcpy(tempComplex, dev_calibFFTShifted, imageSizeL * sizeof(hipfftComplex), hipMemcpyDeviceToHost))
//		cout << "cuda memory cpy error!" << endl;
//	complexWrite("fft after shift", tempComplex, 960, "../Debug/calib_FFT_Shifted.csv");


	hipEvent_t absStart;
	hipEventCreate(&absStart);
	hipEvent_t absStop;
	hipEventCreate(&absStop);
	hipEventRecord(absStart, NULL);

	hipEvent_t absStart;
	hipEventCreate(&absStart);
	hipEvent_t absStop;
	hipEventCreate(&absStop);
	hipEventRecord(absStart, NULL);

	getAbsOfComplexMatric << < gridSize, blockSizeL >> > (dev_calibFFTShifted, dev_calibABSFFTShifted, imageSizeL);


	hipEventRecord(absStop, NULL);
	hipEventSynchronize(absStop);
	float msecABS = 0.0f;
	hipEventElapsedTime(&msecABS	, absStart, absStop);
	cout << "total runtime of ABS: " << msecABS << " ms" << endl;

//	if (hipSuccess != hipGetLastError())
//		printf("get abs error!\n");

	int b = hipMemcpy(calibAbsImage, dev_calibABSFFTShifted, imageSizeL * sizeof(hipfftReal), hipMemcpyDeviceToHost);
//	if (hipSuccess != b)
//		cout << "cuda memory cpy error!" << endl;
//	cout << b << endl;
//	realWrite("calib abs image", calibAbsImage, 960, "../Debug/calib_abs_image.csv");

	calibImage->fftMaxPosition = findMaxPoint(calibAbsImage);
//	cout << "Xmax= " << calibImage->fftMaxPosition.x << ", Ymax= " << calibImage->fftMaxPosition.y << endl;

	hipEvent_t filterStart;
	hipEventCreate(&filterStart);
	hipEvent_t filterStop;
	hipEventCreate(&filterStop);
	hipEventRecord(filterStart, NULL);

	circShift2D << <gridSize, blockSizeL >> > (dev_calibFFTShifted, calibImage->fftMaxPosition, dev_circCalibFFT, imageSizeL);

//	if (hipSuccess != hipGetLastError())
//		printf("circle shift error!\n");
//	if (hipSuccess != hipMemcpy(tempComplex, dev_circCalibFFT, imageSizeL * sizeof(hipfftComplex), hipMemcpyDeviceToHost))
//		cout << "cuda memory cpy error!" << endl;
//	complexWrite("fft after circshift", tempComplex, 960, "../Debug/circFFT.csv");

	createFilter << <gridSize, blockSizeL >> > (80, calibImage->fftMaxPosition, dev_circCalibFFT, dev_calibCircFilteredFFT, imageSizeL);



//	if (hipSuccess != hipGetLastError())
//		printf("filter create error!\n");
//	if (hipSuccess != hipMemcpy(tempComplex, dev_calibCircFilteredFFT, imageSizeL * sizeof(hipfftComplex), hipMemcpyDeviceToHost))
//		cout << "cuda memory cpy error!" << endl;
//	complexWrite("fft after circshift", tempComplex, 960, "../Debug/calib_filtered.csv");

	IFFTShift2D << <gridSize, blockSizeL >> > (dev_calibCircFilteredFFT, dev_filteredCalibFFT, imageSizeL);
//	if (hipSuccess != hipGetLastError())
//		printf("IFFT shift error!\n");

	hipEventRecord(filterStop, NULL);
	hipEventSynchronize(filterStop);
	float msecFilter = 0.0f;
	hipEventElapsedTime(&msecFilter, filterStart, filterStop);
	cout << "total runtime of creating filter: " << msecFilter << " ms" << endl;

//	if (hipSuccess != hipMemcpy(tempComplex, dev_filteredCalibFFT, imageSizeL * sizeof(hipfftComplex), hipMemcpyDeviceToHost))
//		cout << "cuda memory cpy error!" << endl;
//	complexWrite("fft after circshift", tempComplex, 960, "../Debug/ifft_shifted.csv");


	hipEvent_t ifftStart;
	hipEventCreate(&ifftStart);
	hipEvent_t ifftStop;
	hipEventCreate(&ifftStop);
	hipEventRecord(ifftStart, NULL);
	
	hipfftExecC2C(FFT, dev_filteredCalibFFT, dev_calibFilteredBaseband, HIPFFT_BACKWARD);

	hipEventRecord(ifftStop, NULL);
	hipEventSynchronize(ifftStop);
	float msecIFFT = 0.0f;
	hipEventElapsedTime(&msecIFFT, ifftStart, ifftStop);
	cout << "total runtime of FFT: " << msecIFFT << " ms" << endl;

	vectorNumdivide << <gridSize, blockSizeL >> > (dev_calibFilteredBaseband, imageSizeL, imageSizeL);
	if (hipSuccess != hipMemcpy(calibImage->filteredBaseband, dev_calibFilteredBaseband, (calibImage->imagePixels) * sizeof(float2), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;

//	complexWrite("calib filtered baseband", calibImage->filteredBaseband, 960, "../Debug/calib_filtered_baseband.csv");

	if (hipSuccess != hipFree(dev_calibImage))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_calibABSFFTShifted))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_circCalibFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_calibFilteredBaseband))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_calibFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_filteredCalibFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_calibCircFilteredFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_calibFFTShifted))
		cout << "cude memory free error!" << endl;


	hipEventRecord(FFStop, NULL);
	hipEventSynchronize(FFStop);
	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, FFStart, FFStop);
	cout << "total runtime of part fourier filter: " << msecTotal << " ms" << endl;
}

float* phaseRetrieval(image* calibImage, image* testImage) {
	cout << "Part: phase retrieval" << endl;

	int imageSizeL = 1280 * 960;
	dim3 blockSizeL(1, 960, 1), gridSize(1280, 1, 1);
	float2* tempComplex = new float2[imageSizeL];
	float* testAbsImage = (float*)malloc(sizeof(float) * imageSizeL);

	hipfftReal* dev_testABSFFTShifted;
	hipfftComplex* dev_testFFT, * dev_circTestFFT, * dev_testFilteredBaseband, * dev_testCircFilteredFFT, * dev_filteredTestFFT, * dev_testFFTShifted, * dev_testImage;
	int n = hipMalloc((void**)& dev_testImage, sizeof(float2) * testImage->imagePixels);
//	if (hipSuccess != n)
//		cout << "cuda malloc error1!" << endl;
//	cout << n << endl;
	if (hipSuccess != hipMalloc((void**)& dev_testFilteredBaseband, sizeof(float2) * testImage->imagePixels))
		cout << "cuda malloc error2!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_testCircFilteredFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error3!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_testABSFFTShifted, sizeof(float) * imageSizeL))
		cout << "cuda malloc error4!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_testFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error5!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_circTestFFT, sizeof(float2) * testImage->imagePixels))
		cout << "cuda malloc error8!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_filteredTestFFT, sizeof(float2) * testImage->imagePixels))
		cout << "cuda malloc error8!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_testFFTShifted, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error8!" << endl;

	hipfftHandle FFT;
	errorHandle(hipfftPlan2d(&FFT, 1280, 960, HIPFFT_C2C));

	if (hipSuccess != hipMemcpy(dev_testImage, testImage->imageData, testImage->imagePixels * sizeof(float2), hipMemcpyHostToDevice))
		cout << "cuda memory cpy error!" << endl;

	errorHandle(hipfftExecC2C(FFT, dev_testImage, dev_testFFT, HIPFFT_FORWARD));
	FFTShift2D << < gridSize, blockSizeL >> > (dev_testFFT, dev_testFFTShifted, imageSizeL);
//	if (hipSuccess != hipGetLastError())
//		printf("FFT shift Error!\n");

	circShift2D << <gridSize, blockSizeL >> > (dev_testFFTShifted, calibImage->fftMaxPosition, dev_circTestFFT, imageSizeL);
//	if (hipSuccess != hipGetLastError())
//		printf("circ shift Error!\n");
	createFilter << <gridSize, blockSizeL >> > (80, calibImage->fftMaxPosition, dev_circTestFFT, dev_testCircFilteredFFT, imageSizeL);
//	if (hipSuccess != hipGetLastError())
//		printf("filter create Error!\n");
	IFFTShift2D << <gridSize, blockSizeL >> > (dev_testCircFilteredFFT, dev_filteredTestFFT, imageSizeL);
	if (hipSuccess != hipGetLastError())
		printf("IFFT shift Error!\n");
	errorHandle(hipfftExecC2C(FFT, dev_filteredTestFFT, dev_testFilteredBaseband, HIPFFT_BACKWARD));
	vectorNumdivide << <gridSize, blockSizeL >> > (dev_testFilteredBaseband, imageSizeL, imageSizeL);
	if (hipSuccess != hipMemcpy(testImage->filteredBaseband, dev_testFilteredBaseband, (testImage->imagePixels) * sizeof(float2), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error" << endl;
//	complexWrite("test filtered baseband", testImage->filteredBaseband, 960, "../Debug/test_filtered_baseband.csv");

	if (hipSuccess != hipFree(dev_testImage))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_testABSFFTShifted))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_circTestFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_testFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_filteredTestFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_testCircFilteredFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_testFFTShifted))
		cout << "cude memory free error!" << endl;

	float* phaseImage = (float*)malloc(sizeof(float) * imageSizeL);
	
	hipfftReal* dev_phaseImage;
	hipfftComplex* dev_calibFilteredBaseband, * dev_finalImage;
	if (hipSuccess != hipMalloc((void**)& dev_phaseImage, sizeof(float) * imageSizeL))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibFilteredBaseband, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_finalImage, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMemcpy(dev_calibFilteredBaseband, calibImage->filteredBaseband, imageSizeL * sizeof(float2), hipMemcpyHostToDevice))
		cout << "cuda memory cpy error!" << endl;

	vectorMatDivide << <gridSize, blockSizeL >> > (dev_testFilteredBaseband, dev_calibFilteredBaseband, dev_finalImage, calibImage->imagePixels);
	if (hipSuccess != hipGetLastError())
		printf("divide Error!\n");
	phaseCalculate << <gridSize, blockSizeL >> > (dev_finalImage, dev_phaseImage, imageSizeL);
	if (hipSuccess != hipGetLastError())
		printf("phase calculate Error!\n");
	if (hipSuccess != hipMemcpy(phaseImage, dev_phaseImage, imageSizeL * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;
	realWrite("phase image", phaseImage, 960, "../Debug/phase_image1.csv");
	
	float* UnwrappedImage = new float[imageSizeL];

	phaseUnwrapping(phaseImage, UnwrappedImage);

	realWrite("phase image after unwrapping", UnwrappedImage, 1280, "..\ouput_text\phase_image2.csv");
	
	if (hipSuccess != hipMemcpy(dev_phaseImage, phaseImage, testImage->imagePixels * sizeof(float), hipMemcpyHostToDevice))
		cout << "cuda memory cpy error!" << endl;

	float xConf, yConf, xSum = 0, ySum = 0, vecStep, sum2 = 0, mean2;
	for (int i = 0; i < 960; i++) {
		xSum += phaseImage[i * testImage->width + 29] - phaseImage[i * testImage->width + 1279 - 30];
	}
	//for (int i = 0; i < 1280; i++) {
	//	ySum += phaseImage[29 * testImage.width + i] - phaseImage[929 * testImage.width + i];
	//}
	xConf = -1 * (xSum / 960);
	vecStep = xConf / 1279;
	for (int x = 0; x < 100; x++) {
		for (int y = 0; y < 100; y++) {
			sum2 += phaseImage[x + y * testImage->width];
		}
	}
	mean2 = sum2 / 10000;

	hipfftReal* dev_xConfVec;
	if (hipSuccess != hipMalloc((void**)& dev_xConfVec, sizeof(float) * 1280))
		cout << "cuda malloc error!" << endl;
	createXConfVec << <(1, 1, 1), (1280, 1, 1) >> > (xConf, vecStep, dev_xConfVec, 1280);
	if (hipSuccess != hipGetLastError())
		printf("xConf vec create Error!\n");

	forPhaseImage << <gridSize, blockSizeL >> > (mean2, dev_xConfVec, dev_phaseImage, testImage->imagePixels);
	hipFree(dev_xConfVec);
	hipDeviceSynchronize();
	if (hipSuccess != hipMemcpy(phaseImage, dev_phaseImage, testImage->imagePixels * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;
	
	float* height1 = (float*)malloc(sizeof(float) * testImage->imagePixels);
	hipfftReal* dev_height;
	hipMalloc((void**)& dev_height, sizeof(float) * testImage->imagePixels);
	calHeight << <gridSize, blockSizeL >> > (dev_phaseImage, mean2, dev_height, testImage->imagePixels);
	if (hipSuccess != hipGetLastError())
		printf("cal height Error!\n");
	hipDeviceSynchronize();
	if (hipSuccess != hipMemcpy(height1, dev_height, testImage->imagePixels * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;

	float* outputImage = (float*)malloc(sizeof(float) * testImage->imagePixels);
	float* dev_output;
	if (hipSuccess != hipMalloc((void**)& dev_output, sizeof(float) * testImage->imagePixels))
		cout << "cuda malloc error!" << endl;
	calOutputImage << <gridSize, blockSizeL >> > (dev_height, dev_output, testImage->imagePixels);
	if (hipSuccess != hipGetLastError())
		printf("output image create Error!\n");
	hipDeviceSynchronize();
	if (hipSuccess != hipMemcpy(outputImage, dev_output, testImage->imagePixels * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;

	hipFree(dev_output);
	hipFree(dev_height);
	
	return height1;

}

float* phaseUnwrapping(float* wMatrix, float* result) {
	dim3 blockSize(32, 32, 1), gridSize(40, 30, 1);
	int imageSize = 1280 * 960;
	int width = 960;
	int height = 1280;

	float* dev_wMatrix;
	float* dev_GradMatrix, * dev_matrixS, * dev_matrixL, * dev_temp, * dev_unwrapC, * dev_result;
	if (hipSuccess != hipMalloc((void**)& dev_GradMatrix, sizeof(float) * imageSize))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_matrixS, sizeof(double) * imageSize))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_matrixL, sizeof(double) * imageSize))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_temp, sizeof(double) * imageSize))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_unwrapC, sizeof(double) * imageSize))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_unwrapC, sizeof(double) * imageSize))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_wMatrix, sizeof(float) * imageSize))
		cout << "cuda malloc error!" << endl;

	if (hipSuccess != hipMemcpy(dev_wMatrix, wMatrix, 1280 * 960 * sizeof(float), hipMemcpyHostToDevice))
		cout << "cuda memory cpy error!" << endl;
	gradCal << <gridSize, blockSize >> > (dev_wMatrix, dev_GradMatrix, 960, 1280);

	DCTMatrixL<<<gridSize , blockSize >>>(height, dev_matrixS);
	DCTMatrixR<<<gridSize , blockSize >>>(width, dev_matrixL);

	matrixMultiple << <gridSize, blockSize >> > (dev_temp, dev_matrixS, dev_GradMatrix, 960, imageSize);
	matrixMultiple << <gridSize, blockSize >> > (dev_unwrapC, dev_temp, dev_matrixL, 1280, imageSize);
	matrixModify << <gridSize, blockSize >> > (dev_unwrapC, 1280, 960);

	matrixMultiple << <gridSize, blockSize >> > (dev_temp, dev_matrixS, dev_unwrapC, 960, imageSize);
	matrixMultiple << <gridSize, blockSize >> > (dev_result, dev_temp, dev_matrixL, 1280, imageSize);

	if (hipSuccess != hipMemcpy(result, dev_result, imageSize * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;
}

void imageFileWrite(float* input, char* filename) {

	TIFF* tif = TIFFOpen(filename, "w");
	if (tif) {
		TIFFSetField(tif, TIFFTAG_IMAGEWIDTH, 960);
		TIFFSetField(tif, TIFFTAG_IMAGELENGTH, 1280);
		uint8** tempData = new uint8 * [960];
		for (int i = 0; i < 960; i++) {
			tempData[i] = new uint8[1280];
			for (int j = 0; j < 1280; j++) {
				tempData[i][j] = (uint8)input[j + i * 1280];
			}
			TIFFWriteScanline(tif, tempData[i], i);
		}
	}
	else
		cout << filename << " can not be opened!" << endl;

}

void realWrite(const char* title, float* input, int height, const char* filename) {

	ofstream outFile;
	outFile.open(filename);
	outFile.setf(ios::fixed, ios::floatfield);
	outFile.precision(7);
	for (int y = 0; y < height; y++) {
		for (int x = 0; x < 1280; x++) {
			outFile << input[x * height + y] << ",";
		}
		outFile << " " << endl;
	}

}

void complexWrite(const char* title, float2* input, int height, const char* filename) {
	ofstream outFile;
	outFile.open(filename);
	outFile.setf(ios::fixed, ios::floatfield);
	outFile.precision(7);
	for (int y = 0; y < height; y++) {
		for (int x = 0; x < 1280; x++) {
			outFile << input[x * height + y].x << "+" << input[x * height + y].y << "i" << ",";
		}
		outFile << " " << endl;
	}
}

void errorHandle(int input) {
	switch (input)
	{
	case HIPFFT_ALLOC_FAILED:
		cout << "The allocation of GPU resources for the plan failed." << endl;
	case HIPFFT_INVALID_VALUE:
		cout << "One or more invalid parameters were passed to the API." << endl;
	case HIPFFT_INTERNAL_ERROR:
		cout << "An internal driver error was detected." << endl;
	case HIPFFT_SETUP_FAILED:
		cout << "cuFFT library initialize fail." << endl;
	case HIPFFT_INVALID_SIZE:
		cout << "One or more of the nx, ny, or nz parameters is not a supported size." << endl;
	case HIPFFT_INVALID_PLAN:
		cout << "The plan parameter is not a valid handle." << endl;
	case HIPFFT_EXEC_FAILED:
		cout << "cuFFT failed to execute the transform on the GPU." << endl;
	case HIPFFT_SUCCESS:
		break;
	}
}

/*
hipEvent_t start1;
hipEventCreate(&start1);
hipEvent_t stop1;
hipEventCreate(&stop1);
hipEventRecord(start1, NULL);
// ��Ҫ��ʱ����ں˺���kernel;
hipEventRecord(stop1, NULL);
hipEventSynchronize(stop1);
float msecTotal1 = 0.0f;
hipEventElapsedTime(&msecTotal1, start1, stop1);
*/

