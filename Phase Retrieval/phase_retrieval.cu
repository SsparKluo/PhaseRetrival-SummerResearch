#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include"tiffio.h"
#include<cmath>
#include<algorithm>
#include"FT_kernel.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include<hip/hip_runtime.h>
#include"myUnwrap.h"
#include"matrix.h"

#define lambda 0.632
#define pi 3.1415926

typedef unsigned char BYTE;

using namespace std;

class image {
public:
	char filename[50];
	float maxIntensity, minIntensity;
	int height, width;
	int imagePixels;
	uint8** rawImage;
	float* imageData;
	int2 fftMaxPosition;
	float2* filteredBaseband;
	float* imageFilter;

	image() {
		rawImage = (uint8 * *)malloc(sizeof(uint8*) * 960);
		for (int i = 0; i < 960; i++) {
			rawImage[i] = (uint8*)malloc(sizeof(uint8) * 1280);
		}
		filteredBaseband =(float2*)malloc(sizeof(float)*2*1280*960);
		imageFilter = new float[960 * 1280];
	}
};

bool getImageInfo(image* targetImage);// Used for get the useful data in tiff image, and return false when the filename of the target image is invalid.
float* phaseRetrieval(image* calibImage, image* testImage);
void fourierFilterForCalib(image* calibImage);
int2 findMaxPoint(float* input); 
void imageFileWrite(float* input, char* filename);
void complexWrite(const char* title, float2* input, int width, const char* filename);
void realWrite(const char* title, float* input, int width, const char* filename);
void errorHandle(int input);


__global__ void vectorAdd( float* a,  float* b, float* c, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements)
		c[i] = a[i] + b[i];
}

__global__ void vectorNumMultiple( float* a,  float* b, float* c, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements)
		c[i] = a[i] * b[i];
}

__global__ void numMultipleForComplex( float2* a,  float* b, float2* c, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x + threadIdx.y * gridDim.x;
	if (i < numElements) {
		c[i].x = a[i].x * b[i];
		c[i].y = a[i].y * b[i];
	}
}

__global__ void vectorNumdivide( float2* dividend,  float2* divisor, float2* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		output[i].x = dividend[i].x / divisor[i].x;
		output[i].y = dividend[i].y / divisor[i].y;
	}

}

__global__ void getAbsOfComplexMatric( hipfftComplex* input, hipfftReal* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements)
		output[i] = sqrt(input[i].x * input[i].x + input[i].y * input[i].y);
}

__global__ void createFilter( int padding,  int2 maxPoint, float2* input, float2* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		if ((x * x + y * y) <= padding * padding) {
			output[i].x = input[i].x;
			output[i].y = input[i].y;
		}
		else {
			output[i].x = 0;
			output[i].y = 0;
		}
	}
}

__global__ void FFTShift2D( hipfftComplex* input, hipfftComplex* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	int half = gridDim.x / 2;
	if (i < numElements ) {
		if (y >= half / 2) {
			output[(y - half) * gridDim.x + x].x = input[y * gridDim.x + x].x;
			output[(y - half) * gridDim.x + x].y = input[y * gridDim.x + x].y;
		}
		else {
			output[(y + half) * gridDim.x + x].x = input[y * gridDim.x + x].x;
			output[(y + half) * gridDim.x + x].y = input[y * gridDim.x + x].y;
		}
	}
}

__global__ void IFFTShift2D( hipfftComplex* input, hipfftComplex* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	int halfY = blockDim.y / 2;
	int halfX = gridDim.x / 2;
	int preX, preY;
	if (i < numElements) {
		if (y >= halfY)
			preY = y - halfY;
		else
			preY = y + halfY;
		if (x >= halfX)
			preX = x - halfX;
		else
			preX = x + halfX;
		output[x + gridDim.x * y].x = input[preX + gridDim.x * preY].x;
		output[x + gridDim.x * y].y = input[preX + gridDim.x * preY].y;
	}
}

__global__ void circShift2D( hipfftComplex* input,  int2 maxPoint, hipfftComplex* output, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	int preX = x - 640 + maxPoint.x;
	int preY = x - 480 + maxPoint.y;
	if (i < numElements) {
		if (preX < 0)
			preX = 1280 + preX;
		if (preY < 0 || preY > 480) {
			output[x + gridDim.x + y].x = 0;
			output[x + gridDim.x + y].y = 0;
		}
		else {
			output[x + gridDim.x * y].x = input[preX + gridDim.x * preY].x;
			output[x + gridDim.x * y].y = input[preX + gridDim.x * preY].y;
		}
	}
}

__global__ void phaseCalculate( hipfftComplex* input, hipfftReal* output, int numElements) {

	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		output[i] = atan2(input[i].y, input[i].x);
	}
}

__global__ void createXConfVec( float xConf,  float vecStep, hipfftReal* output, int numElements) {
	int i = threadIdx.x;
	if (i < numElements) {
		output[i] = xConf - i * vecStep;
	}
}

__global__ void forPhaseImage( float mean2,  hipfftReal* xConfVec, hipfftReal* input, int numElements) {
	int x = blockIdx.x;
	int y = threadIdx.y;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		input[x + y * gridDim.x] = input[x + y * gridDim.x] + xConfVec[x] - mean2;
		if (input[x + y * gridDim.x] < 0)
			input[x + y * gridDim.x] = -input[x + y * gridDim.x];
	}
}

__global__ void calHeight( hipfftReal* input, float mean2, hipfftReal* output, int numElements) {
	int dn = 0.075;
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		output[i] = (input[i] * lambda / 2 / pi / dn - mean2) / 4 * 3;
	}
}

__global__ void calOutputImage( hipfftReal* input, float* output, int numElements) {
	int i = blockIdx.x * blockDim.y + threadIdx.y;
	if (i < numElements) {
		output[i] = ((input[i] + 15) / 30) * 256 - 1;
	}
}


int main() {
	image calibImage, testImage;
	do
	{
		cout << "please input the filename and the address of the calibration image: ";
		cin >> calibImage.filename;
	} while (!getImageInfo(&calibImage));
	do
	{
		cout << "please input the filename and the address of the test image: ";
		cin >> testImage.filename;
	} while (!getImageInfo(&testImage));
	fourierFilterForCalib(&calibImage);
	float* result = phaseRetrieval(&calibImage,&testImage);
}

bool getImageInfo( image* targetImage ) {
	
	TIFF* tif = TIFFOpen(targetImage->filename, "r");
	if (tif) {
		TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &targetImage->height);
		TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &targetImage->width);
		targetImage->imagePixels = targetImage->height * targetImage->width;
		targetImage->imageData = new float[targetImage->imagePixels];
		targetImage->rawImage = new uint8 * [targetImage->height];
		for (int i = 0; i < targetImage->height; i++) {
			targetImage->rawImage[i] =(uint8*)malloc(sizeof(uint8)*targetImage->width);
		}
		for (int i = 0; i < targetImage->height; i++) {
			TIFFReadScanline(tif, targetImage->rawImage[i], i);
		}
		float tempMax = 0;
		float tempMin = 10000;
		for (int i = 0; i < targetImage->width; i++) {
			for (int j = 0; j < targetImage->height; j++) {
				targetImage->imageData[i*targetImage->height + j] = ((float)targetImage->rawImage[j][i] / 256);
				//if (tempMax < targetImage->imageData[i * targetImage->width + j]) 
				//	tempMax = targetImage->imageData[i * targetImage->width + j];
				//if (tempMin > targetImage->imageData[i * targetImage->width + j])
				//	tempMin = targetImage->imageData[i * targetImage->width + j];
			}
		}
		targetImage->maxIntensity = tempMax;
		targetImage->minIntensity = tempMin;
		TIFFClose(tif);
		return true;
	}
	else {
		cout << "File Open Error! please input a valid filename" << endl;
		return false;
	}
	
	return true;
}

int2 findMaxPoint(float* input) {
	int2 tempPoint = {0,0};
	float tempMax = 0;
	for (int i = 0; i < 481; i++) {
		for (int j = 0; j < 1280; j++) {
			if (j > 600 && j < 680)
				continue;
			if (input[i + j * 481] > tempMax) {
				tempMax = input[i + j * 481];
				tempPoint.y = i;
				tempPoint.x = j;
			}
		}
	}
	return tempPoint;
}

void fourierFilterForCalib(image* calibImage) {
	cout << "Part: fourier filter for calib image" << endl;
	int imageSizeS = 1280 * 481;
	int imageSizeL = 1280 * 960;
	dim3 blockSizeL(1, 960, 1), gridSize(1280, 1, 1), blockSizeS(1, 481, 1);

	float* calibAbsImage = new float[imageSizeS];

	realWrite("input for fourierFiltered", calibImage->imageData, 1280, "../Debug/input_FF.txt");

	hipfftReal* dev_calibImage, * dev_calibABSFFTShifted;
	hipfftComplex* dev_calibFFT, * dev_circCalibFFT, * dev_calibFilteredBaseband, * dev_calibCircFilteredFFT, * dev_filteredCalibFFT;
	int n = hipMalloc((void**)& dev_calibImage, sizeof(float) * calibImage->imagePixels);
	if (hipSuccess != n)
		cout << "cuda malloc error1!" << endl;
	cout << n << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibFilteredBaseband, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error2!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibCircFilteredFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error3!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibABSFFTShifted, sizeof(float2) * imageSizeS))
		cout << "cuda malloc error4!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibFFT, sizeof(float2) * imageSizeS))
		cout << "cuda malloc error5!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_circCalibFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error8!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_filteredCalibFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error8!" << endl;

	hipfftHandle FFT;
	hipfftHandle IFFT;
	errorHandle(hipfftPlan2d(&FFT, 1280, 960, HIPFFT_R2C));
	errorHandle(hipfftPlan2d(&IFFT, 1280, 960, HIPFFT_C2C));

	if (hipSuccess != hipMemcpy(dev_calibImage, calibImage->imageData, calibImage->imagePixels * sizeof(hipfftReal), hipMemcpyHostToDevice))
		cout << "cuda memory cpy error!" << endl;

	errorHandle(hipfftExecR2C(FFT, dev_calibImage, dev_calibFFT));
	float2* tempOut = (float2*)malloc(sizeof(float2) * 1280 * 481);
	int a = hipMemcpy((void*)tempOut, (void*)dev_calibFFT, imageSizeS * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	if (hipSuccess != a)
		cout << "cuda memory cpy error!" << endl;
	cout << a << endl;
	complexWrite("temp dubug info", tempOut, 1280, "../Debug/tempout.txt");

	FFTShift2D <<< gridSize, blockSizeS >>> (dev_calibFFT, dev_calibFFTShifted, imageSizeS);
	if (hipSuccess != hipGetLastError())
		printf("FFTShift error!\n");
	
	getAbsOfComplexMatric <<< gridSize, blockSizeS >>> (dev_calibFFTShifted, dev_calibABSFFTShifted, imageSizeS);
	if (hipSuccess != hipGetLastError())
		printf("get abs error!\n");


	int b = hipMemcpy((void*)calibAbsImage, (void*)dev_calibABSFFTShifted, imageSizeS * sizeof(hipfftReal), hipMemcpyDeviceToHost);
	if (hipSuccess != b)
		cout << "cuda memory cpy error!" << endl;
	cout << b << endl;

	realWrite("calib abs image", calibAbsImage, 640, "..\ouput_text\calib_abs_image.txt");

	calibImage->fftMaxPosition = findMaxPoint(calibAbsImage);

	circShift2D <<<gridSize, blockSizeL >>> (dev_calibFFT, calibImage->fftMaxPosition, dev_circCalibFFT, imageSizeL);
	if (hipSuccess != hipGetLastError())
		printf("circle shift error!\n");
	createFilter << <gridSize, blockSizeL >> > (80, calibImage->fftMaxPosition, dev_circCalibFFT, dev_calibCircFilteredFFT, imageSizeL);//???
	if (hipSuccess != hipGetLastError())
		printf("filter create error!\n");
	IFFTShift2D <<<gridSize, blockSizeL >>> (dev_calibCircFilteredFFT, dev_filteredCalibFFT, imageSizeL);
	if (hipSuccess != hipGetLastError())
		printf("IFFT shift error!\n");
	hipfftExecC2C(IFFT, dev_filteredCalibFFT, dev_calibFilteredBaseband,HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	
	if (hipSuccess != hipMemcpy(calibImage->filteredBaseband, dev_calibFilteredBaseband, (calibImage->imagePixels ) * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;

	complexWrite("calib filtered baseband", calibImage->filteredBaseband, 1280, "..\ouput_text\calib_filtered_baseband.txt");

	if (hipSuccess != hipFree(dev_calibImage))
		cout<<"cude memory free error!"<<endl;
	if (hipSuccess != hipFree(dev_calibABSFFTShifted))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_circCalibFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_calibFilteredBaseband))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_calibFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_filteredCalibFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_calibCircFilteredFFT))
		cout << "cude memory free error!" << endl;

}

float* phaseRetrieval(image* calibImage, image* testImage) {
	cout << "Part: phase retrieval" << endl;
	int imageSizeS = 1280 * 481;
	int imageSizeL = 1280 * 960;
	dim3 blockSizeL(1, 960, 1), gridSize(1280, 1, 1), blockSizeS(1, 641, 1);

	float* testAbsImage = (float*)malloc(sizeof(float) * imageSizeS);

	hipfftReal* dev_testImage, * dev_testABSFFTShifted;
	hipfftComplex* dev_testFFT, * dev_circTestFFT, * dev_testFilteredBaseband, * dev_testCircFilteredFFT, * dev_filteredTestFFT;
	int n = hipMalloc((void**)& dev_testImage, sizeof(float) * testImage->imagePixels);
	if (hipSuccess != n)
		cout << "cuda malloc error1!" << endl;
	cout << n << endl;
	if (hipSuccess != hipMalloc((void**)& dev_testFilteredBaseband, sizeof(float2) * testImage->imagePixels))
		cout << "cuda malloc error2!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_testCircFilteredFFT, sizeof(float2) * imageSizeL))
		cout << "cuda malloc error3!" << endl; 
	if (hipSuccess != hipMalloc((void**)& dev_testABSFFTShifted, sizeof(float2) * imageSizeS))
		cout << "cuda malloc error4!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_testFFT, sizeof(float2) * imageSizeS))
		cout << "cuda malloc error5!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_circTestFFT, sizeof(float2) * testImage->imagePixels))
		cout << "cuda malloc error8!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_filteredTestFFT, sizeof(float2) * testImage->imagePixels))
		cout << "cuda malloc error8!" << endl;

	hipfftHandle FFT;
	errorHandle(hipfftPlan2d(&FFT, 1280, 960, HIPFFT_R2C));
	hipfftHandle IFFT;
	errorHandle(hipfftPlan2d(&IFFT, 1280, 960, HIPFFT_C2C));

	if (hipSuccess != hipMemcpy(dev_testImage, testImage->imageData, testImage->imagePixels * sizeof(float), hipMemcpyHostToDevice))
		cout << "cuda memory cpy error!" << endl;
	//
	errorHandle(hipfftExecR2C(FFT, dev_testImage, dev_testFFT));
	FFTShift2D <<< gridSize, blockSizeS >>> (dev_testFFT, dev_testFFTShifted, imageSizeS);
	if (hipSuccess != hipGetLastError())
		printf("FFT shift Error!\n");
	hipDeviceSynchronize();
	
	circShift2D <<<gridSize, blockSizeL >>> (dev_filteredTestFFT, testImage->fftMaxPosition, dev_circTestFFT, imageSizeL);
	if (hipSuccess != hipGetLastError())
		printf("circ shift Error!\n");
	createFilter << <gridSize, blockSizeL >> > (80, calibImage->fftMaxPosition, dev_circTestFFT, dev_testCircFilteredFFT, imageSizeL);
	if (hipSuccess != hipGetLastError())
		printf("filter create Error!\n");
	IFFTShift2D <<<gridSize, blockSizeL >>> (dev_testCircFilteredFFT, dev_filteredTestFFT, imageSizeL);
	if (hipSuccess != hipGetLastError())
		printf("IFFT shift Error!\n");
	errorHandle(hipfftExecC2C(IFFT, dev_filteredTestFFT, dev_testFilteredBaseband,HIPFFT_BACKWARD));
	hipDeviceSynchronize();
	if (hipSuccess != hipMemcpy(testImage->filteredBaseband, dev_testFilteredBaseband, (testImage->imagePixels) * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error" << endl;

	complexWrite("test filtered baseband", testImage->filteredBaseband, 1280, "..\ouput_text\test_filtered_baseband.txt");

	if (hipSuccess != hipFree(dev_testImage))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_testABSFFTShifted))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_circTestFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_testFilteredBaseband))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_testFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_filteredTestFFT))
		cout << "cude memory free error!" << endl;
	if (hipSuccess != hipFree(dev_testCircFilteredFFT))
		cout << "cude memory free error!" << endl;

	float* phaseImage = (float*)malloc(sizeof(float) * testImage->imagePixels);
	hipfftReal* dev_phaseImage;
	hipfftComplex* dev_calibFilteredBaseband,* dev_finalImage;
	if (hipSuccess != hipMalloc((void**)& dev_phaseImage, sizeof(float) * calibImage->imagePixels))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_calibFilteredBaseband, sizeof(float2) * calibImage->imagePixels))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMalloc((void**)& dev_finalImage, sizeof(float2) * calibImage->imagePixels))
		cout << "cuda malloc error!" << endl;
	if (hipSuccess != hipMemcpy(dev_calibFilteredBaseband, calibImage->filteredBaseband, testImage->imagePixels * sizeof(float2), hipMemcpyHostToDevice))
		cout << "cuda memory cpy error!" << endl;

	vectorNumdivide <<<gridSize,blockSizeL>>> (dev_testFilteredBaseband, dev_calibFilteredBaseband, dev_finalImage, calibImage->imagePixels) ;
	if (hipSuccess != hipGetLastError())
		printf("divide Error!\n");
	phaseCalculate <<<gridSize, blockSizeL >>> (dev_finalImage, dev_phaseImage, calibImage->imagePixels);
	if (hipSuccess != hipGetLastError())
		printf("phase calculate Error!\n");
	if (hipSuccess != hipMemcpy(phaseImage, dev_phaseImage, testImage->imagePixels * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;
	realWrite("phase image", phaseImage, 1280, "..\ouput_text\phase_image1.txt");
	/*
	if (!myUnwrapInitialize()) {
		cout << "matlab unwrap function initialize error" << endl;
	}
	mwArray matlabInput(960, 1280, mxSINGLE_CLASS);
	mwArray matlabOutput(960, 1280, mxSINGLE_CLASS);
	matlabInput.SetData(phaseImage,960*1280);
	myUnwrap(matlabInput);
	matlabOutput.GetData(phaseImage, 960 * 1280);
	//�����˵���matlab\extern�ĵ�ַ
	realWrite("phase image after unwrapping", phaseImage, 1280, "..\ouput_text\phase_image2.txt");
	*/
	if (hipSuccess != hipMemcpy(dev_phaseImage, phaseImage, testImage->imagePixels * sizeof(float), hipMemcpyHostToDevice))
		cout << "cuda memory cpy error!" << endl;

	float xConf, yConf, xSum = 0, ySum = 0, vecStep, sum2=0, mean2;
	for (int i = 0; i < 960; i++) {
		xSum += phaseImage[i * testImage->width + 29] - phaseImage[i * testImage->width + 1279 - 30];
	}
	//for (int i = 0; i < 1280; i++) {
	//	ySum += phaseImage[29 * testImage.width + i] - phaseImage[929 * testImage.width + i];
	//}
	xConf = -1 * (xSum / 960);
	vecStep = xConf / 1279;
	for (int x = 0; x < 100; x++) {
		for (int y = 0; y < 100; y++) {
			sum2 += phaseImage[x + y * testImage->width];
		}
	}
	mean2 = sum2 / 10000;

	hipfftReal* dev_xConfVec;
	if (hipSuccess != hipMalloc((void**)& dev_xConfVec, sizeof(float) * 1280))
		cout << "cuda malloc error!" << endl;
	createXConfVec <<<1,1280 >>> (xConf,vecStep,dev_xConfVec,1280);
	if (hipSuccess != hipGetLastError())
		printf("xConf vec create Error!\n");

	forPhaseImage<<<gridSize,blockSizeL>>>(mean2, dev_xConfVec, dev_phaseImage, testImage->imagePixels);
	hipFree(dev_xConfVec);
	hipDeviceSynchronize();
	if (hipSuccess != hipMemcpy(phaseImage, dev_phaseImage, testImage->imagePixels * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;
	//
	float* height1 = (float*)malloc(sizeof(float) * testImage->imagePixels);
	hipfftReal* dev_height;
	hipMalloc((void**)& dev_height, sizeof(float) * testImage->imagePixels);
	calHeight <<<gridSize, blockSizeL >>> (dev_phaseImage,mean2,dev_height,testImage->imagePixels);
	if (hipSuccess != hipGetLastError())
		printf("cal height Error!\n");
	hipDeviceSynchronize();
	if (hipSuccess != hipMemcpy(height1, dev_height, testImage->imagePixels * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;

	float* outputImage = (float*)malloc(sizeof(float) * testImage->imagePixels);
	float* dev_output;
	if (hipSuccess != hipMalloc((void**)& dev_output, sizeof(float)* testImage->imagePixels))
		cout << "cuda malloc error!" << endl;
	calOutputImage<<<gridSize, blockSizeL >>>(dev_height, dev_output, testImage->imagePixels);
	if (hipSuccess != hipGetLastError())
		printf("output image create Error!\n");
	hipDeviceSynchronize();
	if (hipSuccess != hipMemcpy(outputImage, dev_output, testImage->imagePixels * sizeof(float), hipMemcpyDeviceToHost))
		cout << "cuda memory cpy error!" << endl;

	hipFree(dev_output);
	hipFree(dev_height);

	return height1;
}

void imageFileWrite(float* input, char* filename) {
	/*
	TIFF* tif = TIFFOpen(filename, "w");
	if (tif) {
		TIFFSetField(tif, TIFFTAG_IMAGEWIDTH, 960);
		TIFFSetField(tif, TIFFTAG_IMAGELENGTH, 1280);
		uint8** tempData = new uint8 * [960];
		for (int i = 0; i < 960; i++) {
			tempData[i] = new uint8[1280];
			for (int j = 0; j < 1280; j++) {
				tempData[i][j] = (uint8)input[j + i * 1280];
			}
			TIFFWriteScanline(tif, tempData[i], i);
		}
	}
	else
		cout << filename << " can not be opened!" << endl;
		*/
}

void realWrite(const char* title, float* input , int width, const char* filename) {

	ofstream outFile;
	outFile.open(filename);
	outFile << title << ": " << endl;
	outFile.setf(ios::fixed, ios::floatfield);
	outFile.precision(7);
	outFile << "line 0: ";
	for (int i = 0; i < width; i++) {
		outFile << i << " | ";
	}
	outFile << endl;
	for (int y = 0, int sum = 0; y < 960; y++) {
		outFile << "line " << y << ": ";
		for (int x = 0; x < width; x++) {
			outFile << input[sum] << " | ";
			sum++;
		}
		outFile << endl;
	}

}

void complexWrite(const char* title, float2* input, int width, const char* filename) {
	ofstream outFile;
	outFile.open(filename);
	outFile << title << ": " << endl;
	outFile.setf(ios::fixed, ios::floatfield);
	outFile.precision(7);
	outFile << "line 0: ";
	for (int i = 0; i < width; i++) {
		outFile << i << "|" << i << "i"<< " | ";
	}
	outFile << endl;
	for (int y = 0, int sum = 0; y < 960; y++) {
		outFile << "line " << y << ": ";
		for (int x = 0; x < width; x++) {
			outFile << input[sum].x << "|" << input[sum].y << "i" << " | ";
			sum++;
		}
		outFile << endl;
	}
}

void errorHandle( int input) {
	switch (input)
	{
	case HIPFFT_ALLOC_FAILED :
		cout << "The allocation of GPU resources for the plan failed." << endl;
	case HIPFFT_INVALID_VALUE:
		cout << "One or more invalid parameters were passed to the API." << endl;
	case HIPFFT_INTERNAL_ERROR:
		cout << "An internal driver error was detected." << endl;
	case HIPFFT_SETUP_FAILED:
		cout << "cuFFT library initialize fail." << endl;
	case HIPFFT_INVALID_SIZE:
		cout << "One or more of the nx, ny, or nz parameters is not a supported size." << endl;
	case HIPFFT_INVALID_PLAN:
		cout << "The plan parameter is not a valid handle." << endl;
	case HIPFFT_EXEC_FAILED:
		cout << "cuFFT failed to execute the transform on the GPU." << endl;
	case HIPFFT_SUCCESS:
		cout << "Success" << endl;
	}
}