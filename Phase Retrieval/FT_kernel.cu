#include"FT_kernel.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>


cufffComplex** fourierFilter(double** image, int** imageFilter, int height, int width, int u0, int v0) {

	float* inputData = trans2Dto1D(image, height, width);

	hipfftReal* input = inputData;

	hipfftComplex* imageFFT = FFT();
	hipfftComplex* shiftedFFT;
	hipfftComplex* filterdShiftedFFT;
	hipfftComplex* filterdShiftedFFTBaseband;
	hipfftComplex* filterdFFTBaseband;
	hipfftReal* filterdBaseband;

	hipfftHandle FFTPlan;
	hipfftHandle IFFTPlan;



	hipfftPlan2d(&FFTPlan, height, width, HIPFFT_R2C);
	hipfftPlan2d(&IFFTPlan, height, (width / 2) + 1, HIPFFT_C2R);

}


hipfftComplex* filtering() {

}

 hipfftComplex* fftShift(hipfftComplex* inputFFT, int height, int width) {

}

 

float* trans2Dto1D(double** array2D, int height, int width) {
	float* array1D;
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			array1D[i * width + j] = array2D[i][j];
		}
	}
	return array1D;
}

double** trans1Dto2D(float* array1D, int height, int width) {
	double** array2D;
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			array2D[i][j] = array1D[i * width + j];
		}
	}
	return array2D;
}



